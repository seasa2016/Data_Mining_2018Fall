#include "hip/hip_runtime.h"
#define DATA_SIZE 200000000
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include <stdlib.h>  

using namespace std;

int data[DATA_SIZE];

void GenerateNumber(int *number,int size)
{
    printf("number: ");
    for(int i=0 ; i<size ; i++)
    {
        number[i] = rand()%10;
        //printf("%d ",number[i]);
    }
}

__global__ static void sumOfSquares(int *num,int * result)
{
    //printf("threadIdx.x %d\n",threadIdx.x);
    //printf("blockIdx.x %d\n",blockIdx.x);
    //printf("blockDim.x %d\n",blockDim.x);
    //printf("gridDim.x %d\n",gridDim.x);

    int sum = 0;
    int i;

    for(i=0;i<DATA_SIZE;i++)
        sum += num[i]*num[i];
    *result = sum;
}
__global__ static void sumOfSquares1(int *num,int * result)
{
    const int tid = threadIdx.x;
    const int size = DATA_SIZE / blockDim.x;
    int sum = 0;
    int i;

    for(i=tid*size;i<(tid+1)*size;i++)
        sum += num[i]*num[i];
    result[tid] = sum;
}
__global__ static void sumOfSquares2(int *num,int * result)
{
    const int tid = threadIdx.x;
    int sum = 0;
    int i;

    for(i=tid ; i<DATA_SIZE ; i+= blockDim.x)
        sum += num[i]*num[i];
    result[tid] = sum;
}

__global__ static void sumOfSquares3(int *num,int * result)
{
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int sum = 0;
    int i;

    for(i=bid*blockDim.x+tid ; i<DATA_SIZE ; i+= blockDim.x * gridDim.x)
        sum += num[i]*num[i];
    shared[tid] = sum;

    __syncthreads();
    if(tid==0){
        for(i=1;i<blockDim.x;i++)
            shared[0] += shared[i];
    }
    result[bid] = shared[0];
}


int main()
{
    GenerateNumber(data,DATA_SIZE);
    
    int sum[100] = {0};
    
    clock_t begin_time = clock();
    for(int i=0;i<DATA_SIZE;i++)
        sum[0] += data[i] * data[i];
    printf("time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    printf("sum:%d\n",sum[0]);

    int *gpudata, *result;
    
    

    hipMalloc((void**) &gpudata,sizeof(int)*DATA_SIZE);
    hipMalloc((void**) &result,sizeof(int)*100);
    
    hipMemcpy(gpudata,data,sizeof(int)*DATA_SIZE,hipMemcpyHostToDevice);
    
    begin_time = clock();
    sumOfSquares<<<1,1,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*1*1,hipMemcpyDeviceToHost);
    printf("sum:%d\n",sum[0]);
    printf("---0time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    

    begin_time = clock();
    sumOfSquares1<<<1,100,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*100,hipMemcpyDeviceToHost);
    int s = 0;
    for(int i=0;i<100;i++)
        s += sum[i];
    printf("sum:%d\n",s);
    printf("---1time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    

    begin_time = clock();
    sumOfSquares2<<<1,100,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*100,hipMemcpyDeviceToHost);
    s = 0;
    for(int i=0;i<100;i++)
        s += sum[i];
    printf("sum:%d\n",s);
    printf("---2time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    

    begin_time = clock();
    sumOfSquares3<<<100,100,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*100,hipMemcpyDeviceToHost);
    s = 0;
    for(int i=0;i<100;i++)
        s += sum[i];
    printf("sum:%d\n",s);
    printf("---3time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    



    hipFree(gpudata);
    hipFree(result);
    return 0;
}