#include "hip/hip_runtime.h"
#define DATA_SIZE 200
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include <stdlib.h>  

using namespace std;

int data[DATA_SIZE];

void GenerateNumber(int *number,int size)
{
    printf("number: ");
    for(int i=0 ; i<size ; i++)
    {
        number[i] = rand()%10;
        //printf("%d ",number[i]);
    }
}

__global__ static void sumOfSquares(int *num,int * result,int data)
{
    printf("threadIdx.x %d %d\n",threadIdx.x,data);
    printf("blockIdx.x %d %d\n",blockIdx.x,data);
    printf("blockDim.x %d %d\n",blockDim.x,data);
    printf("gridDim.x %d %d\n",gridDim.x,data);

    int sum = 0;
    int i;

    for(i=0;i<DATA_SIZE;i++)
        sum += num[i]*num[i];
    *result = sum;
}

__global__ static void sumOfSquares1(int *num,int * result)
{
    const int tid = threadIdx.x;
    const int size = DATA_SIZE / blockDim.x;
    int sum = 0;
    int i;

    for(i=tid*size;i<(tid+1)*size;i++)
        sum += num[i]*num[i];
    result[tid] = sum;
}
__global__ static void sumOfSquares2(int *num,int * result)
{
    const int tid = threadIdx.x;
    int sum = 0;
    int i;

    for(i=tid ; i<DATA_SIZE ; i+= blockDim.x)
        sum += num[i]*num[i];
    result[tid] = sum;
}

__global__ static void sumOfSquares3(int *num,int * result)
{
    extern __shared__ int shared[10];
    extern __shared__ int qq[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    printf("size%d\n",sizeof(int));
    printf("threadIdx.x %d %d\n",threadIdx.x);
    printf("threadIdx.y %d %d\n",threadIdx.y);
    printf("blockIdx.x %d %d\n",blockIdx.x);
    printf("blockDim.x %d %d\n",blockDim.x);
    printf("gridDim.x %d %d\n",gridDim.x);

    int sum = 0;
    int i;

    for(i=bid*blockDim.x+tid ; i<DATA_SIZE ; i+= blockDim.x * gridDim.x)
        sum += num[i]*num[i];
    printf("test1\n");

    shared[tid] = sum;
    qq[0] = 0;
    printf("test2\n");


    if(tid==0){
        for(i=1;i<blockDim.x;i++)
            shared[0] += shared[i];
    }
    result[bid] = shared[0];
}

int main()
{
    GenerateNumber(data,DATA_SIZE);
    
    int sum[100] = {0};
    
    clock_t begin_time = clock();
    for(int i=0;i<DATA_SIZE;i++)
        sum[0] += data[i] * data[i];
    printf("time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    printf("sum:%d\n",sum[0]);

    int *gpudata, *result;
    int s;
    

    hipMalloc((void**) &gpudata,sizeof(int)*DATA_SIZE);
    hipMalloc((void**) &result,sizeof(int)*100);
    
    hipMemcpy(gpudata,data,sizeof(int)*DATA_SIZE,hipMemcpyHostToDevice);
    /*
    begin_time = clock();
    sumOfSquares<<<1,3,0>>>(gpudata,result,5);
    hipMemcpy(&sum,result,sizeof(int)*1*1,hipMemcpyDeviceToHost);
    printf("sum:%d\n",sum[0]);
    printf("---0time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    
    
    begin_time = clock();
    sumOfSquares1<<<1,100,0>>>(gpudata,result);
    

    hipMemcpy(&sum,result,sizeof(int)*100,hipMemcpyDeviceToHost);
    s = 0;
    for(int i=0;i<100;i++)
        s += sum[i];
    printf("sum:%d\n",s);
    printf("---1time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    
    while(1);
    begin_time = clock();
    sumOfSquares2<<<1,100,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*100,hipMemcpyDeviceToHost);
    s = 0;
    for(int i=0;i<100;i++)
        s += sum[i];
    printf("sum:%d\n",s);
    printf("---2time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    */

    begin_time = clock();
    sumOfSquares3<<<2,3,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*100,hipMemcpyDeviceToHost);
    s = 0;
    for(int i=0;i<100;i++)
        s += sum[i];
    printf("sum:%d\n",s);
    printf("---3time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    while(true);
        



    hipFree(gpudata);
    hipFree(result);
    return 0;
}