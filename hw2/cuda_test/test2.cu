#include"hip/hip_runtime.h"

#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include <stdlib.h>  

using namespace std;

__device__ int qq(int x)
{
    if(x==0)
        return 0;
    return x+qq(x-1);
}

__global__ static void sumOfSquares3(int * result)
{
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    printf("%d\n",tid*blockDim.x + bid);
    result[tid*blockDim.x + bid] = qq(tid*blockDim.x + bid);
}


int main()
{
    int *result;
    int sum[100] = {0};
    hipMalloc((void**) &result,sizeof(int)*100);
    
    sumOfSquares3<<<10,10,0>>>(result);
    hipMemcpy(&sum,result,sizeof(int)*10*10,hipMemcpyDeviceToHost);
    for(int i=0;i<100;i++)
        printf("-%d\n",sum[i]);
    
    hipFree(result);
    return 0;
}