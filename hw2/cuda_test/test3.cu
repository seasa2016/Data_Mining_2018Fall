#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<thrust/sort.h>
#define LEN 10

__global__ void kernel(int *a, int *b, unsigned int N);

class MyClass {
public:
  MyClass() {
    hipMalloc((void **)&data, sizeof(int)*LEN);
    hipMemset((void *)data, 0, sizeof(int)*LEN);
  };
  
  ~MyClass() {
    hipFree((void *)data);
  };
  
  void run(int *b) {
    dim3 grid(1);
    dim3 block(LEN);
    kernel<<<grid, block>>>(data, b, LEN);
  };
  
  int *get(void) {
    return data;
  };
private:
  int *data;
};

__global__ void kernel(int *a, int *b, unsigned int N) {
  const unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
  if (i<N) {
    a[i] += b[i];
  }
}

void show(int *data, unsigned int N) {
  for (int i=0; i<N; i++) {
    std::cout << data[i] << " ";
  }
  std::cout << std::endl;
}
  
int main(void) {
	/*
	MyClass c;
	int *b_gpu, b_host[LEN];

	for (int i=0; i<LEN; i++) {
		b_host[i] = i;
	}

	hipMalloc((void **)&b_gpu, sizeof(int)*LEN);
	hipMemcpy(b_gpu, b_host, sizeof(int)*LEN, hipMemcpyHostToDevice);
	c.run(b_gpu);
	hipMemcpy(b_host, c.get(), sizeof(int)*LEN, hipMemcpyDeviceToHost);
	hipFree(b_gpu);

	show(b_host, LEN);
	*/
	int test[] = {1,5,3,4,6,7,9,10};
	for(int i=0;i<8;i++)
		printf("%d\n"test[i]);
	thrust::sort(test,test+8);
	for(int i=0;i<8;i++)
		printf("%d\n"test[i]);
}