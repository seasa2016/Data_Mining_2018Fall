
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<thrust/sort.h>

int main(void)
{
	int test[] = {1,5,3,4,6,7,9,10};
	for(int i=0;i<8;i++)
		printf("%d\n",test[i]);
	thrust::sort(test,test+8);
	for(int i=0;i<8;i++)
		printf("%d\n",test[i]);
}