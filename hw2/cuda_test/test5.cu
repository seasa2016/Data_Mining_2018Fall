#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<vector>
#include<thrust/sort.h>
#include<thrust/device_vector.h>


__global__ static void sumOfSquares3(thrust::device_vector<int> * vec)
{
    extern __shared__ int shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    vec->push_back(tid*blockDim.x + bid);
    
}

int main(void)
{
    thrust::device_vector<int> b;
    sumOfSquares3<<<10,10,0>>>(&b);

    std::vector<int> c(b.size());

    thrust::copy(b.begin(),b.end(),c.begin());
    for(int i=0;i<c.size();i++)
        printf("%d\n",c[i]);
}