
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<iostream>
#include <vector>

typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;


__global__ void doThings(Matrix* matrices)
{
    printf("?");
    int i = blockIdx.x;
    for(int j=0;j<5;j++)
        printf("-%f-",matrices[i].elements[j]);
    printf("\n");
    
    matrices[i].elements[0] = 42+i;
    matrices[i].elements[3] = 42+i;
}

int main(void) {
    int rows=5, cols=1, numMat = 16; // These are actually determined at run-time
    Matrix* data = (Matrix*)malloc(numMat * sizeof(Matrix));
    for(int i=0;i<numMat;i++)
    {
        data[i].elements = (float*)malloc(sizeof(float)*5);
        for(int j=0;j<5;j++)
            data[i].elements[j] = j;
    }    
    Matrix* d_data;

    // ... Successfully read from file into "data" ...
    Matrix* h_data = (Matrix*)malloc(numMat * sizeof(Matrix));
    
    hipMalloc(&d_data, numMat*sizeof(Matrix)); 
    hipMemcpy(d_data, data,  numMat * sizeof(Matrix), hipMemcpyHostToDevice);
        
    for (int i=0; i<numMat; i++){
        hipMalloc(&(h_data[i].elements), 5*sizeof(float));
        hipMemcpy(h_data[i].elements, data[i].elements,  5*sizeof(float), hipMemcpyHostToDevice);
        
        hipMemcpy(&(d_data[i].elements), &(h_data[i].elements), sizeof(float*), hipMemcpyHostToDevice);
        
        
    }// matrix data is now on the gpu, now copy the "meta" data to gpu
    
    
    // ... Do other things ...
    doThings<<<numMat,16,0>>>(d_data);

    hipMemcpy(h_data,d_data, numMat*sizeof(Matrix), hipMemcpyDeviceToHost);

    for (int i=0; i<numMat; i++){
        hipMemcpy(data[i].elements,h_data[i].elements,  rows*cols*sizeof(float), hipMemcpyDeviceToHost);
     }// matrix data is now on the gpu, now copy the "meta" data to gpu
     

    for(int i=0;i<numMat;i++)
    {
        for(int j=0;j<5;j++)
            printf("%f ",data[i].elements[j]);
        printf("\n");
    }

}