#include "hip/hip_runtime.h"
#define DATA_SIZE 200000000
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include <stdlib.h>  

using namespace std;

int data[DATA_SIZE];

void GenerateNumber(int *number,int size)
{
    printf("number: ");
    for(int i=0 ; i<size ; i++)
    {
        number[i] = rand()%10;
        //printf("%d ",number[i]);
    }
}


__device__ void qq()
{
    extern __shared__ int shared[][1];
    const int tid = threadIdx.x;
    if(tid==0){
        for(int i=1;i<blockDim.x;i++)
            shared[0][0] += shared[i][0];
    }
}
__global__ static void sumOfSquares(int *num,int * result)
{
    extern __shared__ int shared[][1];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int sum = 0;
    int i;

    for(i=bid*blockDim.x+tid ; i<DATA_SIZE ; i+= blockDim.x * gridDim.x)
        sum += num[i]*num[i];
    shared[tid][0] = sum;
    __syncthreads();

    qq();

    result[bid] = shared[0][0];
}


int main()
{
    GenerateNumber(data,DATA_SIZE);
    
    int sum[100] = {0};
    
    clock_t begin_time = clock();
    for(int i=0;i<DATA_SIZE;i++)
        sum[0] += data[i] * data[i];
    printf("time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    printf("sum:%d\n",sum[0]);

    int *gpudata, *result;
    
    

    hipMalloc((void**) &gpudata,sizeof(int)*DATA_SIZE);
    hipMalloc((void**) &result,sizeof(int)*100);
    
    hipMemcpy(gpudata,data,sizeof(int)*DATA_SIZE,hipMemcpyHostToDevice);
    
    begin_time = clock();
    sumOfSquares<<<10,10,0>>>(gpudata,result);
    hipMemcpy(&sum,result,sizeof(int)*1*1,hipMemcpyDeviceToHost);
    printf("sum:%d\n",sum[0]);
    printf("---0time: %f\n",float( clock () - begin_time ) /  CLOCKS_PER_SEC);
    

    hipFree(gpudata);
    hipFree(result);
    return 0;
}