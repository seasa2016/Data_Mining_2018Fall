#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include<string.h>
#include<vector>
#include<iostream>
#include<bitset>
#include<utility>
#include<algorithm>
#include <time.h>

#define N 524288LL
#define block_size 16
#define thread_size 16

using namespace std;


struct Data{
    int now;
    unsigned int *bit;
};
struct Pre{
    int *now;
    unsigned int *bit;
};
struct stat{
    int *count;
    int *bit;
};
/*
__device__ int my_push_back(Data & pt,int idx) 
{
    int insert_idx = atomicAdd(&count[idx], 1);
    if (insert_idx < N)
    {
        data[idx][insert_idx] = pt;
        return insert_idx;
    }
    else 
        return -1;
}*/

__device__ int bit_count(int i)
{
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}


__device__ void find(Pre *head,int now,Data *data,int i,stat *result,int max,int size,double min_sup)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int total = blockDim.x*gridDim.x;

    extern __shared__ int idx[thread_size];

    int *ori = new int[max];
    int j;
    
    //printf("--at %d %d\n",tid,i);
    for(int j=0;j<max;j++)
    {
        ori[j] = head->bit[j];
    }    
    int count;

    for(;i<size;i++)
    {
        //this result pass the min sup
        //add to the final ans
        count = 0;
        for(j=0;j<max;j++)
        {
            count += bit_count( head->bit[j] & data[i].bit[j]);

        }
        //printf("now: %d i:%d count:%d min_sup:%lf\n",now,i,count,min_sup);
        if( count > min_sup)
        {
            //printf("---%d %d %d\n",tid,now,i);
            head->now[now] = data[i].now;

            for(j=0;j<max;j++)
                head->bit[j] &= data[i].bit[j];
            //printf("---%d %d %d\n",tid,now,i);
                
            //printf("updata %d\n",N/total*(bid*blockDim.x+tid) + idx[tid]);

            //printf("%d\n",tid);
            result[ N/total*(bid*blockDim.x+tid) + idx[tid] ].count[0] = count;
            //printf("%d\n",tid);

            //printf("count %d update %d now: %d\nval ",count,N/total*(bid*blockDim.x+tid) + idx[tid],now);
            for(j=0;j<now+1 ;j++)
            {
                result[ N/total*(bid*blockDim.x+tid) + idx[tid] ].bit[j] = head->now[j];
                printf("%d ",head->now[j]);
            }    
            result[ N/total*(bid*blockDim.x+tid) + idx[tid] ].bit[j] = 0;
            //printf("\n");

            //printf("test i:%d tid:%d idx[tid]:%d\n",i,tid);

            idx[tid] += 1;
            //printf("--%d %d %d\n",tid,now,i);
            
            find(head,now+1,data,i+1,result,max,size,min_sup);

            for(j=0;j<size;j++)
                head->bit[j] = ori[j];
        }
    }
    delete ori;
}

//we should only pass data and result inside here
__global__ static void gpu_find(Data *data,stat *result,int max,int size,double min_sup,int *count)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int total = blockDim.x*gridDim.x;
        
    extern __shared__ int idx[thread_size];

    
    int i,j,k;
    unsigned int temp;
    for(i=0;i<size;i++)
    {
        //printf("%d-",data[i].now);
        for(j=0;j<max;j++)
        {
            temp = data[i].bit[j];
            while(temp)
            {
                //printf("%d",temp%2);
                temp /= 2;
            }
        }
        //printf("\n");
    }

    
    Pre head;
    head.now = new int[size];
    head.bit = new unsigned int[max];
    
    
    for(i=bid*blockDim.x+tid ; i<size ; i+= blockDim.x * gridDim.x)
    {
        //printf("--%d %d i:%d\n",bid,tid,i);
        head.now[0] = data[i].now;
        for(j=0;j<max;j++)
        {
            head.bit[j] = data[i].bit[j];
        }    
        find(&head,1,data,i+1,result,max,size,min_sup);
    }   

    __syncthreads();
    
    //printf("finish\n");
    //move the result to the front of each part
    if(tid == 0)
    {
        count[bid] = 0;
        //for each thread
        for(i=0;i<blockDim.x;i++)
        {
            for(j=0;j<idx[i];j++)
            {
                //printf("(%d,%d)\n",N/gridDim.x*bid + count[bid] + j,N/total*(bid*blockDim.x+i) + j);
                
                result[ N/gridDim.x*bid + count[bid] + j ].count[0] = result[ N/total*(bid*blockDim.x+i) + j ].count[0];
                for(k=0;k<size ;k++)
                    result[ N/gridDim.x*bid + count[bid] + j ].bit[k] = result[ N/total*(bid*blockDim.x+i) + j ].bit[k];
            }

            count[bid] += idx[i];
        }
        //printf("%d count %d\n",bid,count[bid]);
            
    }
}

class ECLAT{
    private:
        vector< pair< int , vector<int> > > input_data;
        double min_sup;
        FILE* output;

        Data *ori_data;
        
        int count,max,size;
    public:
        ECLAT(void){
            
        }
        ECLAT(vector< pair<  int,vector<int> > > &input_data,double min_sup,char* output_file,int max){
            this->min_sup = min_sup;
            
            this->max = (max+31) / 32;
            printf("max: %d\n",this->max);
            this->size = input_data.size();

            this->output = fopen(output_file,"w");

            this->init(input_data);

        }
        void init(vector< pair< int,vector<int> > > &input_data)
        {
            this->ori_data = new Data[input_data.size()];

            //here we first filter out the un sup data
            this->count = 0;
            for(int i=0;i<input_data.size();i++)
            {
                if(input_data[i].second.size()>this->min_sup)
                {
                    this->ori_data[this->count].now = i;
                    this->ori_data[this->count].bit = new unsigned int[this->max];
                    
                    //printf("%d ",this->ori_data[this->count].now);       
                    fprintf(this->output,"%d ",this->ori_data[this->count].now+1);

                    memset(this->ori_data[this->count].bit,0,this->max*sizeof(int));
                    for(int j=0;j<input_data[i].second.size();j++)
                    {
                        this->ori_data[this->count].bit[ input_data[i].second[j] / 32 ] |=  1 << (input_data[i].second[j]%32) ; 

                    }    

                    fprintf(this->output,"(%lu)\n",input_data[i].second.size());
                    //printf("(%lu)\n",input_data[i].second.size());
                    
                    this->count ++;
                }
            }
            /*
            unsigned int temp;
            for(int i=0;i<this->count;i++)
            {
                printf("%d:",this->ori_data[i].now);
                for(int j=0;j<this->max;j++)
                {
                    temp = this->ori_data[i].bit[j];
                    while(temp)
                    {
                        printf("%u",temp%2);
                        temp = temp/2;
                    }
                    printf(" ");
                }    
                printf("\n");
            }
            */
                    
        }
        void freq()
        {
            int i,j,k;
            Data* d_data;
            Data* h_data = (Data*)malloc(this->count * sizeof(Data));
            
            stat* d_result;
            stat* h_result = (stat*)malloc(N * sizeof(stat));

            memcpy(h_data, this->ori_data, this->count * sizeof(Data));
            memset(h_result, 0, N * sizeof(stat));

            //printf("state 1\n");

            for (i=0; i<this->count ; i++){
                hipMalloc(&(h_data[i].bit), this->max*sizeof(int));
                hipMemcpy(h_data[i].bit, this->ori_data[i].bit,  this->max*sizeof(int), hipMemcpyHostToDevice);

            }// matrix data is now on the gpu, now copy the "meta" data to gpu
            for (i=0; i<N ; i++){
                hipMalloc(&(h_result[i].bit), this->size*sizeof(int));
                hipMemset(h_result[i].bit, 0,  this->size*sizeof(int) );
                
                hipMalloc(&(h_result[i].count), sizeof(int));
                hipMemset(h_result[i].count, 0,  sizeof(int));
            }// matrix data is now on the gpu, now copy the "meta" data to gpu

            hipMalloc(&d_data, this->count*sizeof(Data)); 
            hipMemcpy(d_data, h_data, this->count*sizeof(Data), hipMemcpyHostToDevice);
            hipMalloc(&d_result, N*sizeof(stat)); 
            hipMemcpy(d_result, h_result, N*sizeof(stat), hipMemcpyHostToDevice);

            int *d_count;
            int h_count[block_size];
            hipMalloc(&d_count, block_size*sizeof(int)); 
            hipMemset(d_count,0, block_size*sizeof(int));

            //gpu_find(Data *data,stat *result,int max,int size,double min_sup,int *count)
            gpu_find<<<block_size,thread_size,0>>>(d_data,d_result,this->max,this->size,this->min_sup,d_count);
            //printf("after in\n");
            hipMemcpy(h_count,d_count, block_size*sizeof(int), hipMemcpyDeviceToHost);
            //printf("count in\n");
            //while(1);
            
            stat *ans = new stat[N];
            hipMemcpy(h_result , d_result, N*sizeof(stat), hipMemcpyDeviceToHost);
            //printf("----%d\n",N);


            for(i=0;i<N;i++)
            {
                ans[i].bit = new int[this->size];
                ans[i].count = new int[1];
                hipMemcpy(ans[i].bit , h_result[i].bit, this->size*sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(ans[i].count , h_result[i].count, sizeof(int), hipMemcpyDeviceToHost);    

                //printf("bit %d %d %d\n",i,ans[i].bit[0],ans[i].count[0]);
            }
            // matrix data is now on the gpu, now copy the "meta" data to gpu

            
            for(i=0;i<block_size;i++)
            {
                fflush(stdout);
                for(j=0;j<h_count[i];j++)
                {

                    for(k=0;k<this->size;k++)
                    {
                        if( k>0 && ans[ N/block_size*i + j ].bit[k] == 0)
                            break;

                        //printf("%u ",ans[ N/block_size*i + j ].bit[k]+1);       
                        fprintf(this->output,"%u ",ans[ N/block_size*i + j ].bit[k]+1);
                        fflush(stdout);
                    }    
                    fprintf(this->output,"(%d)\n",ans[ N/block_size*i + j ].count[0]);
                    //printf("(%d)\n",ans[ N/block_size*i + j ].count[0]);
                    fflush(stdout);
                }
            }
            
            
            printf("before free\n");
            for (int i=0; i<this->count ; i++)
                hipFree(h_data[i].bit);

            for (int i=0; i<N ; i++)
                hipFree(h_result[i].bit);

            hipFree(d_data); 
            hipFree(d_result); 
            hipFree(d_count);
            printf("finish\n");
            
        }
 
};

int main(int argc,char * argv[])
{
    time_t start, end;

    start = clock();
    int id,i;
    char str[2048];
    char* fir;
    vector< pair< int , vector<int> > > data;

    FILE *in;
    //here we first deal with the input data
    i = 0;
    in = fopen(argv[1],"r");
    while(fgets(str,2048,in))
    {
        fir = strtok(str," ");
        while(fir != NULL)
        {
            sscanf(fir,"%d",&id);
            

            while(data.size()<id)
                data.push_back( make_pair(data.size()+1,vector<int>()));
            
            data[id-1].second.push_back(i);

            fir = strtok(NULL," ");
        }
        i++;
    }
   
    double min_sup;
    sscanf(argv[2],"%lf",&min_sup);

    ECLAT eclat(data,min_sup*i,argv[3],i);
    printf("test1\n");
    eclat.freq();
    printf("test2\n");
    end = clock();

    double diff = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time = %f\n", diff);
}   