#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include"hip/hip_runtime_api.h"
#include<stdio.h>
#include<string.h>
#include<math.h>
#include<vector>
#include<iostream>
#include<bitset>
#include<utility>
#include<algorithm>
#include <time.h>

#define N 1048576
#define block_size 16
#define thread_size 16

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


using namespace std;


struct Data{
    int now;
    unsigned int *bit;
};
struct Pre{
    unsigned int *val;
    unsigned int *bit;
};
struct stat{
    unsigned int *count;
    unsigned int *val;
};


__device__ int my_push_back(stat *result,unsigned int *data,int size,int count,int bid) 
{
    extern __shared__ int idx;
    printf("my push idx:%d bid:%d\n",idx,bid);
    int insert_idx = atomicAdd(&idx, 1);
    if (insert_idx < N/block_size && insert_idx >=0)
    {
        result[ N/block_size*bid + insert_idx ].count[0] = count;

        memcpy(result[  N/block_size*bid + insert_idx ].val,data,size*sizeof(unsigned int));
        
        return insert_idx;
    }
    else 
    {
        printf("error %d %d\n",bid,insert_idx);
        return -1;
    }
}

__device__ int bit_count(unsigned int i)
{
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}


__device__ void find(Pre *head,Data *data,int i,stat *result,int &max,int &size,int &data_size,int min_sup)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    int j;
    int c;


    unsigned int *ori = (unsigned int*)malloc(max*sizeof(unsigned int));
    
    memcpy(ori,head->bit,max*sizeof(unsigned int));
        
    //printf("data_size %d\n",data_size);
    for(;i<data_size;i++)
    {
        //this result pass the min sup
        //add to the final ans
        c = 0;
        for(j=0;j<max;j++)
            c += bit_count( head->bit[j] & data[i].bit[j]);

        printf("(%d,%d) i:%d count:%d min_sup:%d\n",tid,bid,i,c,min_sup);
        if( c > min_sup)
        {
            
            head->val[data[i].now/32] |= (1U << (data[i].now%32));
            for(j=0;j<max;j++)
                head->bit[j] &= data[i].bit[j];
            
            //my_push_back(stat & result,unsigned int *data int size,int count,int bid) 
            my_push_back(result,head->val,size,c,bid);
            
            //find(Pre *head,Data *data,int i,stat *result,int max,int size,int data_size,double min_sup)
            if(1+i<data_size)
                find(head,data,i+1,result,max,size,data_size,min_sup);

            memcpy(head->bit,ori,max*sizeof(unsigned int));

            head->val[data[i].now/32] &= ~(1U << (data[i].now%32));
        }
    }
    free(ori);
}


//we should only pass data and result inside here
__global__ void gpu_find(Data *data,stat *result,int max,int size,int data_size,unsigned int min_sup,int *count)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    
    extern __shared__ int idx;
    idx = 0;
    __syncthreads();
    
    //printf("start:%d %d\n",bid,tid);
    int i,j;
    /*
    if(tid==0 && bid==1)
    {
        for(i=0;i<data_size;i++)
        {
            printf("%d-",data[i].now);
            for(j=0;j<max;j++)
            {
                printf("%u ",data[i].bit[j]);
            }
            printf("\n");
        }
    }
    */
    
    Pre head;
    head.val = new unsigned int[size];
    head.bit = new unsigned int[max];
    memset(head.val,0,size*sizeof(unsigned int));
    
    //printf("size %d max %d data_size %d\n",size,max,data_size);
    for(i=tid*blockDim.x+bid ; i<data_size ; i+= blockDim.x * gridDim.x)
    {
        head.val[data[i].now/32] |= (1U << (data[i].now%32));

        for(j=0;j<max;j++)    
            head.bit[j] = data[i].bit[j];
        
        //find(Pre *head,Data *data,int i,stat *result,int max,int size,int data_size,unsigned int min_sup)
        //find(head,data,i+1,result,max,size,data_size,min_sup);
        
        head.val[data[i].now/32] &= ~(1U << (data[i].now%32));
    }   

    
    __syncthreads();
    
    //printf("finish\n");
    //move the result to the front of each part
    if(tid == 0)
    {
        count[bid] = idx;

        printf("count[%d]:%d\n",bid,count[bid]);
    }
}

class ECLAT{
    private:
        vector< pair< int , vector<int> > > input_data;
        int min_sup;
        FILE* output;

        Data *ori_data;
        
        int count,max,size;
    public:
        ECLAT(void){
            
        }
        ECLAT(vector< pair<  int,vector<int> > > &input_data,double min_sup,char* output_file,int max){
            this->min_sup = (int)(ceil(min_sup));
            printf("this->min_sup %d \n",this->min_sup);
            this->max = (max+31) / 32;
            printf("max: %d\n",this->max);
            this->size = (input_data.size()+31) / 32;
            printf("size: %d\n",this->size);

            this->output = fopen(output_file,"w");

            this->init(input_data);

        }
        void init(vector< pair< int,vector<int> > > &input_data)
        {
            this->ori_data = new Data[input_data.size()];

            //here we first filter out the un sup data
            this->count = 0;
            printf("this->max::%d\n",this->max);
            for(int i=0;i<input_data.size();i++)
            {
                //printf("input_data[%d].second.size() %d\n",i,input_data[i].second.size());
                if(input_data[i].second.size()>this->min_sup)
                {
                    //printf("qq ");
                    this->ori_data[this->count].now = i;
                    this->ori_data[this->count].bit = new unsigned int[this->max];
                    
                    //printf("(%d,%d) ",this->ori_data[this->count].now,input_data[i].second.size());       
                    fprintf(this->output,"%d ",this->ori_data[this->count].now+1);

                    memset(this->ori_data[this->count].bit,0,this->max*sizeof(unsigned int));
                    for(int j=0;j<input_data[i].second.size();j++)
                        this->ori_data[this->count].bit[ input_data[i].second[j] / 32 ] |=  1U << (input_data[i].second[j]%32) ; 

                    fprintf(this->output,"(%lu)\n",input_data[i].second.size());
                    //printf("(%lu)\n",input_data[i].second.size());
                    
                    this->count ++;
                }
            }
            /*
            printf("this->count %d\n",this->count);
            unsigned int temp;
            for(int i=0;i<1;i++)
            {
                printf("%d:\n",i);
                for(int j=0;j<this->max;j++)
                    printf("%u ",this->ori_data[i].bit[j]);
                printf("\n");
            }
            */
        }
        void freq()
        {
            int i,j;
            unsigned int *temp = new unsigned int[this->max];
            memset(temp, 0, this->max * sizeof(unsigned int));
            Data* d_data;
            Data* h_data = new Data[this->count];
            
            stat* d_result;
            stat* h_result = new stat[N];

            memcpy(h_data, this->ori_data, this->count * sizeof(Data));
            memset(h_result, 0, N * sizeof(stat));

            //printf("state 1\n");

            hipMalloc((void**)&d_data, this->count*sizeof(Data)); 
            hipMemcpy(d_data, this->ori_data, this->count*sizeof(Data), hipMemcpyHostToDevice);
            
            for (i=0; i<this->count ; i++){
                hipMalloc((void**)&(h_data[i].bit), this->max*sizeof(unsigned int));
                hipMemcpy(h_data[i].bit, this->ori_data[i].bit,  this->max*sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(&(d_data[i].bit), &(h_data[i].bit),  sizeof(unsigned int*), hipMemcpyHostToDevice);
            }// matrix data is now on the gpu, now copy the "meta" data to gpu
            

            hipMalloc((void**)&d_result, N*sizeof(stat)); 
            for (i=0; i<N ; i++){
                hipMalloc((void**)&(h_result[i].val), this->size*sizeof(unsigned int));
                hipMemset(h_result[i].val, 0,  this->size*sizeof(unsigned int) );
                hipMemcpy(&(d_result[i].val), &(h_result[i].val),  sizeof(unsigned int*), hipMemcpyHostToDevice);
                
                hipMalloc((void**)&(h_result[i].count), sizeof(unsigned int));
                hipMemset(h_result[i].count, 0,  sizeof(unsigned int));
                hipMemcpy(&(d_result[i].count), &(h_result[i].count),  sizeof(unsigned int*), hipMemcpyHostToDevice);
            }// matrix data is now on the gpu, now copy the "meta" data to gpu

            

            int *d_count;
            int *h_count = new int[block_size];
            hipMalloc((void**)&d_count, block_size*sizeof(int)); 
            hipMemset(d_count,0, block_size*sizeof(int));

            printf("this->count:%d\n",this->count);
            //gpu_find(Data *data,stat *result,int max,int size,int data_size,unsigned int min_sup,int *count)
            gpu_find<<<block_size,thread_size>>>(d_data,d_result,this->max,this->size,this->count,this->min_sup,d_count);
            fflush(stdout);
            hipDeviceSynchronize();
            cudaCheckErrors("???????????????");
            
            hipMemcpy(h_count,d_count, block_size*sizeof(unsigned int), hipMemcpyDeviceToHost);
            
            stat *ans = new stat[N];
            hipMemcpy(h_result , d_result, N*sizeof(stat), hipMemcpyDeviceToHost);

            for(i=0;i<N;i++)
            {
                ans[i].val = new unsigned int[this->size];
                ans[i].count = new unsigned int[1];

                hipMemcpy(ans[i].val , h_result[i].val, this->size*sizeof(unsigned int), hipMemcpyDeviceToHost);
                hipMemcpy(ans[i].count , h_result[i].count, sizeof(unsigned int), hipMemcpyDeviceToHost);    

                //printf("bit %d %d %d\n",i,ans[i].bit[0],ans[i].count[0]);
            }
            // matrix data is now on the gpu, now copy the "meta" data to gpu

            unsigned int k;
            for(i=0;i<block_size;i++)
            {
                printf("h_count[%d] %d\n",i,h_count[i]);    
                fflush(stdout);   
                for(j=0;j<h_count[i];j++)
                {

                    for(k=0;k<this->size*32;k++)
                    {
                        if( (1U<<(k%32)) & ans[ N/block_size*i + j ].val[k/32])
                        {
                            //printf("%u ",k+1);       
                            fprintf(this->output,"%u ",k+1);
                            fflush(stdout);
                        }
                    }    
                    //printf("(%u)\n",ans[ N/block_size*i + j ].count[0]);
                    fprintf(this->output,"(%u)\n",ans[ N/block_size*i + j ].count[0]);
                    fflush(stdout);
                }
            }
            
            
            printf("before free\n");
            for (int i=0; i<this->count ; i++)
                hipFree(h_data[i].bit);

            for (int i=0; i<N ; i++)
                hipFree(h_result[i].val);

            hipFree(d_data); 
            hipFree(d_result); 
            hipFree(d_count);
            printf("finish\n");
            
        }
 
};

int main(int argc,char * argv[])
{
    time_t start, end;

    start = clock();
    int id,i;
    char str[2048];
    char* fir;
    vector< pair< int , vector<int> > > data;

    FILE *in;
    //here we first deal with the input data
    i = 0;
    in = fopen(argv[1],"r");
    while(fgets(str,2048,in))
    {
        fir = strtok(str," ");
        while(fir != NULL)
        {
            sscanf(fir,"%d",&id);
            

            while(data.size()<id)
                data.push_back( make_pair(data.size()+1,vector<int>()));
            
            data[id-1].second.push_back(i);

            fir = strtok(NULL," ");
        }
        i++;
    }
   
    double min_sup;
    sscanf(argv[2],"%lf",&min_sup);

    ECLAT eclat(data,min_sup*i,argv[3],i);
    printf("test1\n");
    eclat.freq();
    printf("test2\n");
    end = clock();

    double diff = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time = %f\n", diff);
}   