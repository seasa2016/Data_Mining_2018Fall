
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<vector>
#include<iostream>
#include<utility>
#include<algorithm>
#include <time.h>
#include<math.h>

#define block_size 256
#define thread_size 256
using namespace std;


__device__ int bit_count(unsigned int i)
{
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}
__global__ void gpu_inter(unsigned int * query,unsigned int** bank,unsigned int** d_result,int *d_count,int start,int max,int size)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i,j;
	
    //move the query on the the sharded memory

    //use parella to compute all the result
    for(i=bid + start ; i<size ; i+= gridDim.x) 
        for(j=tid ; j<max ; j+=blockDim.x)
        	d_result[i][j] = query[j] & bank[i][j];
    
    
	__syncthreads();

    for(i=bid + gridDim.x*tid + start ; i<size ; i+= gridDim.x*blockDim.x)
    {	
		d_count[i] = 0;
		for(j=0;j<max;j++)
			d_count[i] += bit_count(d_result[i][j]);
	}
}   


class ECLAT{
    public:
        int min_sup;
        FILE* output;
        unsigned int **d_data;
        unsigned int **h_data;

        unsigned int **data;

        unsigned int **h_result;
        unsigned int **d_result;

        unsigned int *d_query;

        int *h_count;
        int *d_count;

        int *pre;

        int max,size;
		vector<unsigned int**> result;
		vector<int*> count;

        ECLAT(void){
            
        }
        ECLAT(vector< pair< int , vector<int> > > &input_data,int max,double min_sup,char* output_file){
            this->min_sup = int(ceil(min_sup));
            
            this->output = fopen(output_file,"w");
            
            this->max = (max+31)/32;

            this->init(input_data);
        }
        void print(unsigned *x)
        {
            for(int j=0;j<this->max*32;j++)
            {
                if(x[j/32] & 1UL<<(j%32)) printf("1");
                else printf("0");
                
                if(j && j%32==0) printf(" ");
            }
            printf("\n");
            fflush(stdout);
        }
        void init(vector< pair< int , vector<int> > > &input_data)
        {
            //here we first filter out the un sup data
            
            vector< pair<int,unsigned int*> > data_temp;
            
            //finst parsing the data
            for(int i=0;i<input_data.size();i++)
                if(input_data[i].second.size()>=this->min_sup)
                {
                    unsigned int *temp = new unsigned int[this->max];
                    
                    memset(temp,0,this->max*sizeof(int));

                    for(int j=0;j<input_data[i].second.size();j++)
                        temp[input_data[i].second[j]/32] |= (1UL << (input_data[i].second[j]%32));
                    
                    data_temp.push_back( make_pair(input_data[i].first,temp) );
                }
            //put the data into cpu memory
            this->size = data_temp.size();

            this->pre = new int[data_temp.size()];
            this->data = new unsigned int*[data_temp.size()];

            for(int i=0 ; i<data_temp.size() ; i++)
            {
                this->pre[i] = data_temp[i].first;
                this->data[i] = data_temp[i].second;
				//printf("%3d:",this->pre[i]);
				//print(this->data[i]);
            }

            //we should alloc all the memory first XD    
            this->h_data  = new unsigned int*[data_temp.size()];
            this->h_result  = new unsigned int*[data_temp.size()];

            //alloc memory to 2d array
            hipMalloc(&(this->d_data), data_temp.size()*sizeof(unsigned int*)); 
            hipMalloc(&(this->d_result), data_temp.size()*sizeof(unsigned int*)); 

            hipMemcpy(this->d_data, this->data,  data_temp.size()*sizeof(unsigned int*), hipMemcpyHostToDevice);
            printf("finish 2d\n"); 
            for(int i=0; i<data_temp.size(); i++){
                //alloc memory to 1d array
                hipMalloc(&(this->h_data[i]), (this->max)*sizeof(unsigned int));
                hipMalloc(&(this->h_result[i]), (this->max)*sizeof(unsigned int));

                hipMemcpy(this->h_data[i], this->data[i],  (this->max)*sizeof(unsigned int) , hipMemcpyHostToDevice);
                
                hipMemcpy(&(this->d_data[i]), &(this->h_data[i]), sizeof(unsigned int*), hipMemcpyHostToDevice);
                hipMemcpy(&(this->d_result[i]), &(this->h_result[i]), sizeof(unsigned int*), hipMemcpyHostToDevice);
            }
            printf("finish 1d\n"); 

            hipMalloc((void**)&(this->d_query), this->max*sizeof(unsigned int));
            hipMalloc((void**)&(this->d_count), this->size* sizeof(int));  
			
            printf("max:%d this->size:%d\n",this->max,this->size);
        }
        // use_gpu( bit , now, result, h_count);
        void use_gpu( unsigned int *query,int now,unsigned int**result,int* h_count)
        {
            //we only copy the data here
            hipMemcpy(this->d_query, query, this->max*sizeof(unsigned int), hipMemcpyHostToDevice);

			//printf("start count\n");
            gpu_inter<<<block_size,thread_size,0>>>(this->d_query,this->d_data,this->d_result,this->d_count,now,this->max,this->size);
            
			hipDeviceSynchronize();
			//printf("finish count\n");
            //move result and count back to the cpu
            hipMemcpy(this->h_result,this->d_result, this->size*sizeof(unsigned int*), hipMemcpyDeviceToHost);
            for (int i=now; i<this->size; i++)
                hipMemcpy(result[i],this->h_result[i],  this->max*sizeof(unsigned int), hipMemcpyDeviceToHost);

            hipMemcpy(h_count,this->d_count, this->size*sizeof(int), hipMemcpyDeviceToHost);
        }

        void find(vector<int> &arr,int idx, unsigned int* bit,int now)
        {
            int i;
            int* count_temp;
            unsigned int** result_temp;

			//printf("idx:%d now %d\n",idx,now);
            while(arr.size()<=idx)
			{
				arr.push_back(0);

            	count_temp = new int[this->size];
            	result_temp = new unsigned int*[this->size];
            	for(i=0;i<this->size;i++)
                	result_temp[i] = new unsigned int[this->max];
				
				this->result.push_back(result_temp);
				this->count.push_back(count_temp);
			}

			//printf("use gpu\n");
            use_gpu( bit , now, this->result[idx], this->count[idx]);
           
			/*
			printf("query:  ");
			print(bit);
			for(i=0;i<this->size;i++)
            {
				printf("%3d %3d:",h_count[i],this->pre[i]);
				print(result[i]);
			}
			*/
            for(;now<this->size;now++)
            {
				//printf("in\n");
                //since we share the memory    
                if( this->count[idx][now] >= this->min_sup)
                {
                    arr[idx] = this->pre[now];
                    for(i=0 ; i<idx+1 ; i++)
                        fprintf(this->output,"%d ",arr[i]+1);
                    fprintf(this->output,"(%d)\n",this->count[idx][now]);

                    find(arr,idx+1,this->result[idx][now],now+1);
                }
				//printf("out\n");
            }
			
        }
        void finish()
        {
            hipFree(this->d_query);
            hipFree(this->d_count);  

			int i,j;
            for(i=0; i<this->size; i++){
                hipFree(this->h_data[i]);
                hipFree(this->h_result[i]);
            }

            hipFree(this->d_data); 
            hipFree(this->d_result);

			
			for(i=0;i<this->result.size();i++)
			{
				for(j=0;j<this->size;j++)
					delete(this->result[i][j]);

				delete(this->result[i]);
				delete(this->count[i]);
			}
        }
};

int main(int argc,char * argv[])
{
    time_t start, end;

    start = clock();
    int id,i;
    char str[4096];
    char* fir;
    vector< pair< int , vector<int> > > data;

    FILE *in;
    //here we first deal with the input data
    i = 0;
    in = fopen(argv[1],"r");
    int max = 0;

    printf("parsing data\n");
    while(fgets(str,4096,in))
    {
        fir = strtok(str," ");
        while(fir != NULL)
        {
            sscanf(fir,"%d",&id);
            while(data.size()<id)
                data.push_back( make_pair(data.size(),vector<int>()));
            
            data[id-1].second.push_back(i);
            if(max<i)
                max=i;

            fir = strtok(NULL," ");
        }
        i++;
    }
   
    double min_sup;
    sscanf(argv[2],"%lf",&min_sup);
    printf("initial\n");
    ECLAT eclat(data,max,min_sup*i,argv[3]);

    pair< vector<int> , unsigned int* > head;

    head.first.clear();
    head.second = new unsigned int[eclat.max];

    printf("eclat.max %d\n",eclat.max);
    for(int i=0;i<eclat.max;i++)
        head.second[i] = 0xFFFFFFFF;
    //cout << head.second;

    printf("find freq\n");
    eclat.find(head.first,0,head.second,0);
	printf("finish");
    eclat.finish();
    delete(head.second);

    end = clock();


    double diff = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time = %f\n", diff);
}   
