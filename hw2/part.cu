
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<vector>
#include<iostream>
#include<utility>
#include<algorithm>
#include <time.h>
#include<math.h>

#define block_size 16
#define thread_size 256
using namespace std;


__device__ int bit_count(unsigned int i)
{
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}
__global__ void gpu_inter(unsigned int * query,unsigned int** bank,unsigned int** d_result,int *d_count,int start,int max,int size)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    extern __shared__ unsigned int q[];
    int i,j;
    //move the query on the the sharded memory
    if(tid==0)
        for(i=0;i<max;i++)
            q[i] = query[i];
    __syncthreads();

    //use parella to compute all the result
    for(i=bid*blockDim.x+tid + start ; i<size ; i+= blockDim.x * gridDim.x)
    {
        d_count[i] = 0;
        
        for(j=0;j<max;j++)
        {
            d_result[i][j] = q[j] & bank[i][j];

            d_count[i] += bit_count(d_result[i][j]);
        }
    }
}   


class ECLAT{
    public:
        int min_sup;
        FILE* output;
        unsigned int **d_data;
        unsigned int **h_data;

        unsigned int **data;

        unsigned int **h_result;
        unsigned int **d_result;

        unsigned int *d_query;

        int *h_count;
        int *d_count;

        int *pre;

        int max,size;
        ECLAT(void){
            
        }
        ECLAT(vector< pair< int , vector<int> > > &input_data,int max,double min_sup,char* output_file){
            this->min_sup = int(ceil(min_sup));
            
            this->output = fopen(output_file,"w");
            
            this->max = (max+31)/32;

            this->init(input_data);
        }
        void print(unsigned *x)
        {
            for(int j=0;j<this->max*32;j++)
            {
                if(x[j/32] & 1UL<<(j%32)) printf("1");
                else printf("0");
                
                if(j && j%32==0) printf(" ");
            }
            printf("\n");
            fflush(stdout);
        }
        void init(vector< pair< int , vector<int> > > &input_data)
        {
            //here we first filter out the un sup data
            
            vector< pair<int,unsigned int*> > data_temp;
            
            //finst parsing the data
            for(int i=0;i<input_data.size();i++)
                if(input_data[i].second.size()>=this->min_sup)
                {
                    unsigned int *temp = new unsigned int[this->max];
                    
                    memset(temp,0,this->max*sizeof(int));

                    for(int j=0;j<input_data[i].second.size();j++)
                        temp[input_data[i].second[j]/32] |= (1UL << (input_data[i].second[j]%32));
                    
                    data_temp.push_back( make_pair(input_data[i].first,temp) );
                }
            //put the data into cpu memory
            this->size = data_temp.size();

            this->pre = new int[data_temp.size()];
            this->data = new unsigned int*[data_temp.size()];

            for(int i=0 ; i<data_temp.size() ; i++)
            {
                this->pre[i] = data_temp[i].first;
                this->data[i] = data_temp[i].second;
            }

            //we should alloc all the memory first XD    
            this->h_data  = new unsigned int*[data_temp.size()];
            this->h_result  = new unsigned int*[data_temp.size()];

            //alloc memory to 2d array
            hipMalloc(&(this->d_data), data_temp.size()*sizeof(unsigned int*)); 
            hipMalloc(&(this->d_result), data_temp.size()*sizeof(unsigned int*)); 

            hipMemcpy(this->d_data, this->data,  data_temp.size()*sizeof(unsigned int*), hipMemcpyHostToDevice);
                
            for(int i=0; i<data_temp.size(); i++){
                //alloc memory to 1d array
                hipMalloc(&(this->h_data[i]), (this->max)*sizeof(unsigned int));
                hipMalloc(&(this->h_result[i]), (this->max)*sizeof(unsigned int));

                hipMemcpy(this->h_data[i], this->data[i],  (this->max)*sizeof(unsigned int) , hipMemcpyHostToDevice);
                
                hipMemcpy(&(this->d_data[i]), &(this->h_data[i]), sizeof(unsigned int*), hipMemcpyHostToDevice);
                hipMemcpy(&(this->d_result[i]), &(this->h_result[i]), sizeof(unsigned int*), hipMemcpyHostToDevice);
            }

            hipMalloc((void**)&(this->d_query), this->max*sizeof(unsigned int));
            hipMalloc((void**)&(this->d_count), this->size* sizeof(int));  

            printf("max:%d this->size:%d",this->max,this->size);
        }
        // use_gpu( bit , now, result, h_count);
        void use_gpu( unsigned int *query,int now,unsigned int**result,int* h_count)
        {
            //we only copy the data here
            hipMemcpy(this->d_query, query, this->max*sizeof(unsigned int), hipMemcpyHostToDevice);

            // gpu_inter(unsigned int * query,unsigned int** bank,unsigned int** d_result,int *d_count,int start,int max,int size)
            gpu_inter<<<block_size,thread_size,0>>>(this->d_query,this->d_data,this->d_result,this->d_count,now,this->max,this->size);
            
            //move result and count back to the cpu
            hipMemcpy(this->h_result,this->d_result, this->size*sizeof(unsigned int*), hipMemcpyDeviceToHost);
            for (int i=0; i<this->size; i++)
                hipMemcpy(result[i],this->h_result[i],  this->size*sizeof(unsigned int), hipMemcpyDeviceToHost);

            hipMemcpy(h_count,this->d_count, this->size*sizeof(int), hipMemcpyDeviceToHost);
        }

        void find(vector<int> &arr,int idx, unsigned int* bit,int now)
        {
            unsigned int** result;
            int* h_count;
            int i;

            while(arr.size()<=idx)
                arr.push_back(0);

            h_count = new int[this->size];
            result = new unsigned int*[this->size];
            for(i=0;i<this->size;i++)
                result[i] = new unsigned int[this->size];

            use_gpu( bit , now, result, h_count);
            
            for(;now<this->size;now++)
            {
                //since we share the memory    
                if( h_count[now] >= this->min_sup)
                {
                    arr[idx] = this->pre[now];
                    for(i=0 ; i<idx+1 ; i++)
                        fprintf(this->output,"%d ",arr[i]+1);
                    fprintf(this->output,"(%d)\n",h_count[now]);

                    find(arr,idx+1,result[now],now+1);
                }
                delete(result[i]);
            }
            delete(result);
            delete(h_count);
        }
        void finish()
        {
            hipFree(this->d_query);
            hipFree(this->d_count);  

            for(int i=0; i<this->size; i++){
                hipFree(this->h_data[i]);
                hipFree(this->h_result[i]);
            }

            hipFree(this->d_data); 
            hipFree(this->d_result); 
        }
};

int main(int argc,char * argv[])
{
    time_t start, end;

    start = clock();
    int id,i;
    char str[4096];
    char* fir;
    vector< pair< int , vector<int> > > data;

    FILE *in;
    //here we first deal with the input data
    i = 0;
    in = fopen(argv[1],"r");
    int max = 0;

    printf("parsing data\n");
    while(fgets(str,4096,in))
    {
        fir = strtok(str," ");
        while(fir != NULL)
        {
            sscanf(fir,"%d",&id);
            while(data.size()<id)
                data.push_back( make_pair(data.size(),vector<int>()));
            
            data[id-1].second.push_back(i);
            if(max<i)
                max=i;

            fir = strtok(NULL," ");
        }
        i++;
    }
   
    double min_sup;
    sscanf(argv[2],"%lf",&min_sup);
    printf("initial\n");
    ECLAT eclat(data,max,min_sup*i,argv[3]);

    pair< vector<int> , unsigned int* > head;

    head.first.clear();
    head.second = new unsigned int[eclat.max];

    printf("eclat.max %d\n",eclat.max);
    for(int i=0;i<eclat.max;i++)
        head.second[i] = 0xFFFFFFFF;
    //cout << head.second;

    printf("find freq\n");
    eclat.find(head.first,0,head.second,0);

    eclat.finish();
    delete(head.second);

    end = clock();


    double diff = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time = %f\n", diff);
}   
