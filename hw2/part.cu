
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<vector>
#include<iostream>
#include<utility>
#include<algorithm>
#include <time.h>
#include<math.h>

#define block_size 16
#define thread_size 16
using namespace std;


__device__ int bit_count(unsigned int i)
{
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}
__global__ void gpu_inter(unsigned int * d_vec_x,unsigned int * d_vec_y,unsigned int * d_result,int *d_count,int max)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ int sum[thread_size];
    int i;

    sum[tid] = 0;
    for(i=bid*blockDim.x+tid ; i<max ; i+= blockDim.x * gridDim.x)
    {
        d_result[i] = d_vec_x[i] & d_vec_y[i];
        sum[tid] += bit_count(d_result[i]);
    }

    __syncthreads();
    
    if(tid == 0)
    {
        d_count[bid] = 0;
        for(i=0;i<blockDim.x;i++)
            d_count[bid] += sum[i];
    }    
}   


class ECLAT{
    public:
        vector< pair<int, unsigned int*> > input_data;
        int min_sup;
        char * output_file;
        vector< pair<vector<int>,int> > ans;

        int max,size;
        ECLAT(void){
            
        }
        ECLAT(vector< pair< int , vector<int> > > &input_data,int max,double min_sup,char* output_file){
            this->min_sup = int(ceil(min_sup));
            this->output_file = output_file;
            this->ans.clear();
            
            this->max = (max+31)/32;

            this->init(input_data);
        }
        void init(vector< pair< int , vector<int> > > &input_data)
        {
            //here we first filter out the un sup data
            for(int i=0;i<input_data.size();i++)
                if(input_data[i].second.size()>=this->min_sup)
                {
                    unsigned int *temp = new unsigned int[this->max];
                    
                    memset(temp,0,this->max*sizeof(int));

                    for(int j=0;j<input_data[i].second.size();j++)
                        temp[input_data[i].second[j]/32] |= (1UL << (input_data[i].second[j]%32));
                    
                    this->input_data.push_back( make_pair(input_data[i].first,temp) );
                }
            printf("max:%d this->input_data:%d",this->max,this->input_data.size());
        }
        pair<int, unsigned int*> use_gpu( unsigned int *x , unsigned int *y)
        {
            unsigned int *d_vec_x;
            unsigned int *d_vec_y;

            unsigned int *d_result;
            int *d_count;

            //move to gpu
            hipMalloc((void**)&d_vec_x, this->max*sizeof(unsigned int)); 
            hipMemcpy(d_vec_x, x, this->max*sizeof(unsigned int), hipMemcpyHostToDevice);

            hipMalloc((void**)&d_vec_y, this->max*sizeof(unsigned int)); 
            hipMemcpy(d_vec_y, y, this->max*sizeof(unsigned int), hipMemcpyHostToDevice);
            
            hipMalloc((void**)&d_result, this->max*sizeof(unsigned int)); 
            hipMemset(d_result, 0, this->max*sizeof(unsigned int));

            hipMalloc((void**)&d_count,block_size* sizeof(int)); 
            hipMemset(d_result, 0, block_size*sizeof(int));


            gpu_inter<<<block_size,thread_size>>>(d_vec_x,d_vec_y,d_result,d_count,this->max);
            
            //move to cpu
            unsigned int *h_result = new unsigned int[this->max];
            int *h_count = new int[block_size];


            hipMemcpy(h_result,d_result, this->max*sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(h_count,d_count, block_size*sizeof(int), hipMemcpyDeviceToHost);

            int count = 0;
            for(int i=0;i<block_size;i++)
                count += h_count[i];

            hipFree(d_result);
            hipFree(d_vec_x);
            hipFree(d_vec_y);
            hipFree(d_count);

            delete(h_count);

            return pair<int, unsigned int*>(count,h_result);
        }

        void find(vector<int> arr, unsigned int* bit,int now)
        {
            pair<int, unsigned int*> result;
            
            int idx = arr.size();
            arr.push_back(0);

            for(int i = now;i<this->input_data.size();i++)
            {
                //this result pass the min sup
                //add to the final ans

                result = use_gpu( bit , this->input_data[i].second);
                if( result.first >= this->min_sup)
                {
                    arr[idx] = this->input_data[i].first;
                    
                    this->ans.push_back(pair< vector<int>,int>(arr,result.first));
                    
                    this->find(arr,result.second,i+1);
                }
                delete(result.second);
            }
        }

        void freq(vector<int> arr, unsigned int* bit)
        {
            this->find(arr,bit,0);

            FILE* output = fopen(this->output_file,"w");
            //here we finish find the ans 
            //next we need to do the sort for the ans
            for(int i=0 ; i<this->ans.size() ; i++)
            {
                sort(this->ans[i].first.begin(),this->ans[i].first.end());
                for(int j=0;j<this->ans[i].first.size();j++)
                    fprintf(output,"%d ",this->ans[i].first[j]+1);
                fprintf(output,"(%d)\n",this->ans[i].second);
            }
        }
};

int main(int argc,char * argv[])
{
    time_t start, end;

    start = clock();
    int id,i;
    char str[4096];
    char* fir;
    vector< pair< int , vector<int> > > data;

    FILE *in;
    //here we first deal with the input data
    i = 0;
    in = fopen(argv[1],"r");
    int max = 0;

    printf("parsing data\n");
    while(fgets(str,4096,in))
    {
        fir = strtok(str," ");
        while(fir != NULL)
        {
            sscanf(fir,"%d",&id);
            while(data.size()<id)
                data.push_back( make_pair(data.size(),vector<int>()));
            
            data[id-1].second.push_back(i);
            if(max<i)
                max=i;

            fir = strtok(NULL," ");
        }
        i++;
    }
   
    double min_sup;
    sscanf(argv[2],"%lf",&min_sup);
    printf("initial\n");
    ECLAT eclat(data,max,min_sup*i,argv[3]);

    pair< vector<int> , unsigned int* > head;

    head.first.clear();
    head.second = new unsigned int[eclat.max];

    memset(head.second,1,eclat.max*sizeof(unsigned int));
    //cout << head.second;

    printf("find freq\n");
    eclat.freq(head.first,head.second);
    delete(head.second);

    end = clock();


    double diff = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time = %f\n", diff);
}   