
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<vector>
#include<iostream>
#include<utility>
#include<algorithm>
#include <time.h>
#include<math.h>

#define block_size 16
#define thread_size 16
using namespace std;


__device__ int bit_count(unsigned int i)
{
    i = i - ((i >> 1) & 0x55555555);
    i = (i & 0x33333333) + ((i >> 2) & 0x33333333);
    return (((i + (i >> 4)) & 0x0F0F0F0F) * 0x01010101) >> 24;
}
__global__ void gpu_inter(unsigned int * d_vec_x,unsigned int * d_vec_y,unsigned int * d_result,int *d_count,int max)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    __shared__ int sum[thread_size];
    int i;

    sum[tid] = 0;
    for(i=bid*blockDim.x+tid ; i<max ; i+= blockDim.x * gridDim.x)
    {
        d_result[i] = d_vec_x[i] & d_vec_y[i];
        sum[tid] += bit_count(d_result[i]);
    }

    __syncthreads();
    
    if(tid == 0)
    {
        d_count[bid] = 0;
        for(i=0;i<blockDim.x;i++)
            d_count[bid] += sum[i];
    }    
}   


class ECLAT{
    public:
        vector< pair<int, unsigned int*> > input_data;
        int min_sup;
        FILE* output;
        vector< pair<vector<int>,int> > ans;

        int max,size;
        ECLAT(void){
            
        }
        ECLAT(vector< pair< int , vector<int> > > &input_data,int max,double min_sup,char* output_file){
            this->min_sup = int(ceil(min_sup));
            
            this->output = fopen(output_file,"w");
            
            this->ans.clear();
            
            this->max = (max+31)/32;

            this->init(input_data);
        }
        void print(unsigned *x)
        {
            for(int j=0;j<this->max*32;j++)
            {
                if(x[j/32] & 1UL<<(j%32)) printf("1");
                else printf("0");
                
                if(j && j%32==0) printf(" ");
            }
            printf("\n");
            fflush(stdout);
        }
        void init(vector< pair< int , vector<int> > > &input_data)
        {
            //here we first filter out the un sup data
            unsigned int qq;
            for(int i=0;i<input_data.size();i++)
                if(input_data[i].second.size()>=this->min_sup)
                {
                    unsigned int *temp = new unsigned int[this->max];
                    
                    memset(temp,0,this->max*sizeof(int));

                    //printf("(%d %d)\n",input_data[i].first,input_data[i].second.size());
                    for(int j=0;j<input_data[i].second.size();j++)
                        temp[input_data[i].second[j]/32] |= (1UL << (input_data[i].second[j]%32));
                    
                    //printf("%3d:",input_data[i].first);
                    //print(temp);

                    this->input_data.push_back( make_pair(input_data[i].first,temp) );
                }
            printf("max:%d this->input_data:%d",this->max,this->input_data.size());
        }
        pair<int, unsigned int*> use_gpu( unsigned int *x , unsigned int *y)
        {
            unsigned int *d_vec_x;
            unsigned int *d_vec_y;

            unsigned int *d_result;
            int *d_count;

            //print(x);
            //print(y);
            

            //move to gpu
            hipMalloc((void**)&d_vec_x, this->max*sizeof(unsigned int)); 
            hipMemcpy(d_vec_x, x, this->max*sizeof(unsigned int), hipMemcpyHostToDevice);

            hipMalloc((void**)&d_vec_y, this->max*sizeof(unsigned int)); 
            hipMemcpy(d_vec_y, y, this->max*sizeof(unsigned int), hipMemcpyHostToDevice);
            
            hipMalloc((void**)&d_result, this->max*sizeof(unsigned int)); 
            hipMemset(d_result, 0, this->max*sizeof(unsigned int));

            hipMalloc((void**)&d_count,block_size* sizeof(int)); 
            hipMemset(d_count, 0, block_size*sizeof(int));


            gpu_inter<<<block_size,thread_size,0>>>(d_vec_x,d_vec_y,d_result,d_count,this->max);
            
            //move to cpu
            unsigned int *h_result = new unsigned int[this->max];
            int *h_count = new int[block_size];


            hipMemcpy(h_result,d_result, this->max*sizeof(unsigned int), hipMemcpyDeviceToHost);
            hipMemcpy(h_count,d_count, block_size*sizeof(int), hipMemcpyDeviceToHost);

            int count = 0;
            for(int i=0;i<block_size;i++)
                count += h_count[i];
                    

            //count = count/0;
            hipFree(d_result);
            hipFree(d_vec_x);
            hipFree(d_vec_y);
            hipFree(d_count);

            delete(h_count);

            return pair<int, unsigned int*>(count,h_result);
        }

        void find(vector<int> &arr,int idx, unsigned int* bit,int now)
        {
            pair<int, unsigned int*> result;
            
            while(arr.size()<=idx)
                arr.push_back(0);

            for(;now<this->input_data.size();now++)
            {
                //this result pass the min sup
                //add to the final ans

                result = use_gpu( bit , this->input_data[now].second);
                if( result.first >= this->min_sup)
                {
                    //printf("(%d %d %d)\n",idx,i,result.first);
                    arr[idx] = this->input_data[now].first;
                    
                    //this->ans.push_back(pair< vector<int>,int>(arr,result.first));
                    
                    for(int i=0 ; i<idx+1 ; i++)
                        fprintf(output,"%d ",arr[i]+1);
                    fprintf(output,"(%d)\n",result.first);
                    
                    this->find(arr,idx+1,result.second,now+1);
                }
                delete(result.second);
            }
        }

};

int main(int argc,char * argv[])
{
    time_t start, end;

    start = clock();
    int id,i;
    char str[4096];
    char* fir;
    vector< pair< int , vector<int> > > data;

    FILE *in;
    //here we first deal with the input data
    i = 0;
    in = fopen(argv[1],"r");
    int max = 0;

    printf("parsing data\n");
    while(fgets(str,4096,in))
    {
        fir = strtok(str," ");
        while(fir != NULL)
        {
            sscanf(fir,"%d",&id);
            while(data.size()<id)
                data.push_back( make_pair(data.size(),vector<int>()));
            
            data[id-1].second.push_back(i);
            if(max<i)
                max=i;

            fir = strtok(NULL," ");
        }
        i++;
    }
   
    double min_sup;
    sscanf(argv[2],"%lf",&min_sup);
    printf("initial\n");
    ECLAT eclat(data,max,min_sup*i,argv[3]);

    pair< vector<int> , unsigned int* > head;

    head.first.clear();
    head.second = new unsigned int[eclat.max];

    printf("eclat.max %d\n",eclat.max);
    for(int i=0;i<eclat.max;i++)
        head.second[i] = 0xFFFFFFFF;
    //cout << head.second;

    printf("find freq\n");
    eclat.find(head.first,0,head.second,0);
    delete(head.second);

    end = clock();


    double diff = ((double) (end - start)) / CLOCKS_PER_SEC;

    printf("Time = %f\n", diff);
}   